#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <cstring>
#include <ctime>
#include <fstream>
#include <functional>
#include <iostream>
#include <limits>
#include <list>
#include <map>
#include <queue>
#include <set>
#include <stack>
#include <string>
#include <vector>

#define FLANN_USE_CUDA
#include <flann/flann.hpp>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>

#include <gsl/gsl_blas.h>
#include <gsl/gsl_linalg.h>
#include <gsl/gsl_math.h>
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_vector.h>

using namespace std;

bool file_read(string filename,vector<vector<float>>& vec)
{
	ifstream file(filename);
	if(file.is_open())
	{
		string line;
		getline(file,line);
		int n_line=stoi(line);
		vec=vector<vector<float>>(n_line, vector<float>(3));
		for(int i=0;i<n_line;i++)
		{
			getline(file,line);
			string::size_type sz_x,sz_y,sz_z;
			vec[i][0]=std::stod(line,&sz_x);
			vec[i][1]=std::stod(line.substr(sz_x),&sz_y);
			vec[i][2]=std::stod(line.substr(sz_x).substr(sz_y),&sz_z);
			std::stod(line.substr(sz_x).substr(sz_y).substr(sz_z));
		}
		file.close();
		return true;
	}
	else
	{
		return false;
	}
}

int main( int argc, char** argv)
{
	if(argc != 4)
	{
		cout<<"usage: ./icp file1 file2 no_of_iterations\n";
	}
	else
	{
		int nn = 1;	// No. of nearest neighbours to search
		vector<vector<float>> reading, reference;	// reading and reference point clouds
		if(!file_read(argv[1], reading))
		{
			exit(1);
		}
		if(!file_read(argv[2], reference))
		{
			exit(1);
		}
		
		thrust::host_vector<float4> query_host(reading.size());
		thrust::host_vector<float4> data_host(reference.size());	// cuda code
		
		for(int i=0; i<reading.size(); ++i)	// copy point cloud data
		{
			query_host[i]=make_float4(reading[i][0], reading[i][1], reading[i][2], 0);	// cuda code
		}
		for(int i=0; i<reference.size(); ++i)
		{
			data_host[i]=make_float4(reference[i][0], reference[i][1], reference[i][2], 0);	// cuda code
		}
		
		thrust::device_vector<float4> query_device = query_host;
		thrust::device_vector<float4> data_device = data_host;	// cuda code
		
		flann::Matrix<float> query_device_matrix( (float*)thrust::raw_pointer_cast(&query_device[0]),reading.size(),3,4*4);	// cuda code
		flann::Matrix<float> data_device_matrix( (float*)thrust::raw_pointer_cast(&data_device[0]),reference.size(),3,4*4);
				
		flann::KDTreeCuda3dIndexParams index_params;	// cuda code
		index_params["input_is_gpu_float4"]=true;
		flann::KDTreeCuda3dIndex<flann::L2_Simple<float> > index(data_device_matrix, index_params);
    index.buildIndex();

		thrust::device_vector<int> indices_device(reading.size()*nn);	// cuda code
		thrust::device_vector<float> dists_device(reading.size()*nn);
		flann::Matrix<int> indices_device_matrix( (int*)thrust::raw_pointer_cast(&indices_device[0]),reading.size(),nn);
	flann::Matrix<float> dists_device_matrix( (float*)thrust::raw_pointer_cast(&dists_device[0]),reading.size(),nn);
		
		flann::SearchParams sp;
		sp.matrices_in_gpu_ram=true;
				
		int iterations = stoi(argv[3]);	// No. of icp iterations
		
		flann::Matrix<int> indices_host( new int[ reading.size()*nn],reading.size(),nn );	// cuda code
    flann::Matrix<float> dists_host( new float[ reading.size()*nn],reading.size(),nn );
		
		gsl_matrix *TF = gsl_matrix_alloc(4, 4);
		gsl_vector *query_mean = gsl_vector_alloc(3);
		gsl_vector *dataset_mean = gsl_vector_alloc(3);
		gsl_matrix *U = gsl_matrix_alloc(3, 3);
		gsl_matrix *V = gsl_matrix_alloc(3, 3);
		gsl_vector *S = gsl_vector_alloc(3);
		gsl_vector *work = gsl_vector_alloc(3);
		gsl_matrix *R = gsl_matrix_alloc(3, 3);
		gsl_vector *t = gsl_vector_alloc(3);
		gsl_matrix *T = gsl_matrix_alloc(4, 4);
		gsl_vector *temp1 = gsl_vector_alloc(3);
		gsl_matrix *temp2 = gsl_matrix_alloc(4, 4);
		gsl_vector *temp3 = gsl_vector_alloc(4);
		gsl_vector *temp4 = gsl_vector_alloc(4);
		
		gsl_matrix_set_identity(TF);
		
		for(int i=0; i<iterations; ++i)
		{
	    index.knnSearch(query_device_matrix, indices_device_matrix, dists_device_matrix, nn, sp );	//cuda code
	    
    	thrust::copy( dists_device.begin(), dists_device.end(), dists_host.ptr() );	// cuda code
			thrust::copy( indices_device.begin(), indices_device.end(), indices_host.ptr() );
			
			gsl_vector_set_zero(query_mean);
			for(int j=0; j<reading.size(); ++j)
			{
				gsl_vector_set(query_mean, 0, gsl_vector_get(query_mean, 0)+query_host[j].x/reading.size());
				gsl_vector_set(query_mean, 1, gsl_vector_get(query_mean, 1)+query_host[j].y/reading.size());
				gsl_vector_set(query_mean, 2, gsl_vector_get(query_mean, 2)+query_host[j].z/reading.size());
			}
			
			gsl_vector_set_zero(dataset_mean);
			for(int j=0; j<reference.size(); ++j)
			{
				gsl_vector_set(dataset_mean, 0, gsl_vector_get(dataset_mean, 0)+data_host[j].x/reading.size());
				gsl_vector_set(dataset_mean, 1, gsl_vector_get(dataset_mean, 1)+data_host[j].y/reading.size());
				gsl_vector_set(dataset_mean, 2, gsl_vector_get(dataset_mean, 2)+data_host[j].z/reading.size());
			}
			
			gsl_matrix_set_zero(U);
			for(int j=0; j<reading.size(); ++j)
			{
				gsl_matrix_set(U, 0, 0, gsl_matrix_get(U, 0, 0)+(data_host[indices_host[j][0]].x-gsl_vector_get(dataset_mean, 0))*(query_host[j].x-gsl_vector_get(query_mean, 0)));
				gsl_matrix_set(U, 0, 1, gsl_matrix_get(U, 0, 1)+(data_host[indices_host[j][0]].x-gsl_vector_get(dataset_mean, 0))*(query_host[j].y-gsl_vector_get(query_mean, 1)));
				gsl_matrix_set(U, 0, 2, gsl_matrix_get(U, 0, 2)+(data_host[indices_host[j][0]].x-gsl_vector_get(dataset_mean, 0))*(query_host[j].z-gsl_vector_get(query_mean, 2)));
				gsl_matrix_set(U, 1, 0, gsl_matrix_get(U, 1, 0)+(data_host[indices_host[j][0]].y-gsl_vector_get(dataset_mean, 1))*(query_host[j].x-gsl_vector_get(query_mean, 0)));
				gsl_matrix_set(U, 1, 1, gsl_matrix_get(U, 1, 1)+(data_host[indices_host[j][0]].y-gsl_vector_get(dataset_mean, 1))*(query_host[j].y-gsl_vector_get(query_mean, 1)));
				gsl_matrix_set(U, 1, 2, gsl_matrix_get(U, 1, 2)+(data_host[indices_host[j][0]].y-gsl_vector_get(dataset_mean, 1))*(query_host[j].z-gsl_vector_get(query_mean, 2)));
				gsl_matrix_set(U, 2, 0, gsl_matrix_get(U, 2, 0)+(data_host[indices_host[j][0]].z-gsl_vector_get(dataset_mean, 2))*(query_host[j].x-gsl_vector_get(query_mean, 0)));
				gsl_matrix_set(U, 2, 1, gsl_matrix_get(U, 2, 1)+(data_host[indices_host[j][0]].z-gsl_vector_get(dataset_mean, 2))*(query_host[j].y-gsl_vector_get(query_mean, 1)));
				gsl_matrix_set(U, 2, 2, gsl_matrix_get(U, 2, 2)+(data_host[indices_host[j][0]].z-gsl_vector_get(dataset_mean, 2))*(query_host[j].z-gsl_vector_get(query_mean, 2)));
			}
				
			
			gsl_linalg_SV_decomp(U, V, S, work);
			gsl_blas_dgemm(CblasNoTrans, CblasTrans, 1.0, U, V, 0.0, R);

			gsl_blas_dgemv(CblasNoTrans, 1.0, R, query_mean, 0.0, temp1);
			gsl_vector_memcpy(t, dataset_mean);
			gsl_vector_sub(t, temp1);
			
			gsl_matrix_set_identity(T);
			for(int j=0; j<3; ++j)
			{
				for(int k=0; k<3; k++)
				{
					gsl_matrix_set(T, j, k, gsl_matrix_get(R, j, k));
				}
				gsl_matrix_set(T, j, 3, gsl_vector_get(t, j));
			}
			
			for(int j=0; j<reading.size(); ++j)
			{
				gsl_vector_set(temp3, 0, query_host[j].x);
				gsl_vector_set(temp3, 1, query_host[j].y);
				gsl_vector_set(temp3, 2, query_host[j].z);
				gsl_vector_set(temp3, 3, 1.0);

				gsl_blas_dgemv(CblasNoTrans, 1.0, T, temp3, 0.0, temp4);

				query_host[j].x = gsl_vector_get(temp4, 0);
				query_host[j].y = gsl_vector_get(temp4, 1);
				query_host[j].z = gsl_vector_get(temp4, 2);
			}
			
			gsl_blas_dgemm(CblasNoTrans, CblasNoTrans, 1.0, T, TF, 0.0, temp2);
			gsl_matrix_memcpy(TF, temp2);
			/*
			cout<<endl<<"iteration - "<<i+1<<endl;
			for(int j=0; j<4; ++j)
			{
				for(int k=0; k<4; ++k)
				{
					cout<<gsl_matrix_get(TF, j, k)<<"     ";
				}
				cout<<endl;
			}
			*/
			
/*			
			for(int j=0; j<reading.size(); ++j)
			{
				cout<<query_host[j].x<<"  "<<query_host[j].y<<"  "<<query_host[j].z<<"  "<<query_host[j].w<<"  -  "<<indices_host[j][0]<<"  -  "<<data_host[indices_host[j][0]].x<<"  "<<data_host[indices_host[j][0]].y<<"  "<<data_host[indices_host[j][0]].z<<"  "<<data_host[indices_host[j][0]].w<<"  -  "<<dists_host[j][0]<<endl;
			}
			*/
		}
	}
	return 0;
}
