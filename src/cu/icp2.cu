#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <cstring>
#include <ctime>
#include <fstream>
#include <functional>
#include <iostream>
#include <limits>
#include <list>
#include <map>
#include <queue>
#include <set>
#include <stack>
#include <string>
#include <vector>

#define FLANN_USE_CUDA
#include <flann/flann.hpp>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>

#include <gsl/gsl_blas.h>
#include <gsl/gsl_linalg.h>
#include <gsl/gsl_math.h>
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_vector.h>

using namespace std;

bool file_read(string filename,vector<vector<float>>& vec)
{
	ifstream file(filename);
	if(file.is_open())
	{
		string line;
		getline(file,line);
		int n_line=stoi(line);
		vec=vector<vector<float>>(n_line, vector<float>(3));
		for(int i=0;i<n_line;i++)
		{
			getline(file,line);
			string::size_type sz_x,sz_y,sz_z;
			vec[i][0]=std::stod(line,&sz_x);
			vec[i][1]=std::stod(line.substr(sz_x),&sz_y);
			vec[i][2]=std::stod(line.substr(sz_x).substr(sz_y),&sz_z);
			std::stod(line.substr(sz_x).substr(sz_y).substr(sz_z));
		}
		file.close();
		return true;
	}
	else
	{
		return false;
	}
}

int main( int argc, char** argv)
{
	if(argc != 4)
	{
		cout<<"usage: ./icp file1 file2 no_of_iterations\n";
	}
	else
	{
		int nn = 1;	// No. of nearest neighbours to search
		vector<vector<float>> reading, reference;	// reading and reference point clouds
		if(!file_read(argv[1], reading))
		{
			exit(1);
		}
		if(!file_read(argv[2], reference))
		{
			exit(1);
		}
		
		thrust::host_vector<float4> query_host(reading.size());
		thrust::host_vector<float4> data_host(reference.size());	// cuda code
				
		for(int i=0; i<reading.size(); ++i)	// copy point cloud data
		{
			query_host[i]=make_float4(reading[i][0], reading[i][1], reading[i][2], 0);	// cuda code
		}
		for(int i=0; i<reference.size(); ++i)
		{
			data_host[i]=make_float4(reference[i][0], reference[i][1], reference[i][2], 0);	// cuda code
		}
		
		thrust::device_vector<float4> query_device = query_host;
		thrust::device_vector<float4> data_device = data_host;	// cuda code
		
		flann::Matrix<float> query_device_matrix( (float*)thrust::raw_pointer_cast(&query_device[0]),reading.size(),3,4*4);	// cuda code
		flann::Matrix<float> data_device_matrix( (float*)thrust::raw_pointer_cast(&data_device[0]),reference.size(),3,4*4);
				
		flann::KDTreeCuda3dIndexParams index_params;	// cuda code
		index_params["input_is_gpu_float4"]=true;
		flann::KDTreeCuda3dIndex<flann::L2_Simple<float> > index(data_device_matrix, index_params);
    index.buildIndex();

		thrust::device_vector<int> indices_device(reading.size()*4);	// cuda code
		thrust::device_vector<float> dists_device(reading.size()*4);
		flann::Matrix<int> indices_device_matrix( (int*)thrust::raw_pointer_cast(&indices_device[0]),reading.size(),4);
	flann::Matrix<float> dists_device_matrix( (float*)thrust::raw_pointer_cast(&dists_device[0]),reading.size(),4);
		
		flann::SearchParams sp;
		sp.matrices_in_gpu_ram=true;
				
		int iterations = stoi(argv[3]);	// No. of icp iterations
		
		flann::Matrix<int> indices_host( new int[ reading.size()*4],reading.size(),4 );	// cuda code
    flann::Matrix<float> dists_host( new float[ reading.size()*4],reading.size(),4 );
				
		for(int i=0; i<iterations; ++i)
		{
	    index.knnSearch(query_device_matrix, indices_device_matrix, dists_device_matrix, nn, sp );	//cuda code
	    
    	thrust::copy( dists_device.begin(), dists_device.end(), dists_host.ptr() );	// cuda code
			thrust::copy( indices_device.begin(), indices_device.end(), indices_host.ptr() );
			
			for(int j=0; j<reading.size(); ++j)
			{
//				cout<<query_host[j].x<<"  "<<query_host[j].y<<"  "<<query_host[j].z<<"  -  "<<indices_host[j][0]<<"  -  "<<data_host[indices_host[j][0]].x<<"  "<<data_host[indices_host[j][0]].y<<"  "<<data_host[indices_host[j][0]].z<<"  -  "<<dists_host[j][0]<<endl;
			}
		}
	}
	return 0;
}
